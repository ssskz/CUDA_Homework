#include "hip/hip_runtime.h"
#include<string.h>
#include<math.h>
#include<stdlib.h>
#include<stdio.h>
#define N 100

//GPU VectorAddition Function
__global__ void vecAdd(float* A,float* B,float* C){
    int i=threadIdx.x;
    if(i<N)
        C[i]=A[i]+B[i];
}

int main(){
    //Some initializations of the elements.
    size_t size = N * sizeof(float);
    float *h_A, *h_B, *h_C, *h_D;
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    h_D = (float*)malloc(size);
    float* d_A;
    hipMalloc((void**)&d_A, size);
    float* d_B;
    hipMalloc((void**)&d_B, size);
    float* d_C;
    hipMalloc((void**)&d_C, size);
    srand(time(NULL));
    for(int i=0;i<N;i++){
        h_A[i] = rand()%100;
        h_B[i] = rand()%100;
    }
  
    //Get the CPU running time, using cudaEvent method
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for(int i=0;i<N;i++){
        h_D[i] = h_A[i] + h_B[i];
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Processing time: %f (ms)\n", elapsedTime);
  
    //Use the GPU to check whether the addition is right.
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int threadsPerGrid = (N + threadsPerBlock-1)/threadsPerBlock;
    vecAdd<<<threadsPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  
    //Check the result of the vector addition.
    for(int i = 0; i < N; i++){
        if(h_C[i] = h_D[i]){
            ;
        }
        else{
            printf("Erro! The vector addition is wrong!\n");
            return -1;
        }
    }
    printf("The vector addition is right!\n");
  
    //Free the space of the vector.
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
