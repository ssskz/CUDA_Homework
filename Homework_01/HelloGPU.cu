#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloGPU(void)
{
	printf("thread_no: %d  Hello GPU!\n",threadIdx.x);
}
int main()
{
	//hello from CPU
	printf("Hello CPU!\n");
	helloGPU <<<2, 10 >>> ();
	hipDeviceReset();
	return 0;
}
