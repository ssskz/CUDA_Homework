#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

void InitData(float* data, size_t nElem)
{
	for (size_t i = 0; i < nElem; i++)
	{
		data[i] = i % 255;
	}
}

void SumArraysOnHost(float* h_A, float* h_B, float* hostRef, size_t nElem)
{
	for (size_t i = 0; i < nElem; i++)
	{
		hostRef[i] = h_A[i] + h_B[i];
	}
}

void CheckResults(float* hostRef, float* gpuRef, size_t nElem)
{
	bool bSame = true;
	for (size_t i = 0; i < nElem; i++)
	{
		if (abs(gpuRef[i] - hostRef[i]) > 1e-5)
		{
			bSame = false;
		}
	}

	if (bSame)
	{
		printf("Result is correct!\n");
	}
	else
	{
		printf("Result is error!\n");
	}
}

__global__ void GpuSumArrays(float* d_A, float* d_B, float* d_C, size_t nElem)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < nElem)
		d_C[tid] = d_A[tid] + d_B[tid];
}

int main()
{
	int nDev = 0;
	hipSetDevice(nDev);

	hipDeviceProp_t stDeviceProp;
	hipGetDeviceProperties(&stDeviceProp, nDev);

	hipGetDevice(&nDev);

	//check whether support mapped memory
	if (!stDeviceProp.canMapHostMemory)
	{
		printf("Device %d does not support mapping CPU host memory!\n", nDev);
		goto EXIT;
	}

	printf("Using device %d: %s\n", nDev, stDeviceProp.name);

	// set up data size of vector
	int nPower = 10;
	int nElem = 1 << nPower;
	size_t nBytes = nElem * sizeof(float);
	if (nPower < 18) {
		printf("Vector size %d power %d nbytes %3.0f KB\n",
			nElem, nPower, (float)nBytes / (1024.0f));
	}
	else {
		printf("Vector size %d power %d nbytes %3.0f MB\n",
			nElem, nPower, (float)nBytes / (1024.0f * 1024.0f));
	}

	// part2: using UMA - managed memory for array A , B, C
	// allocate UMA memory
	hipError_t err = hipMallocManaged(&h_A, nBytes);
	if (err != hipSuccess)
	{
		printf("Not support hipMallocManaged!\n");
		goto EXIT;
	}

	hipMallocManaged(&h_B, nBytes, hipMemAttachGlobal);
	hipMallocManaged(&h_C, nBytes, hipMemAttachGlobal);


	// initialize data at host side
	InitData(h_A, nElem);
	InitData(h_B, nElem);
	InitData(h_C, nElem);
	memset(hostRef, 0, nBytes);
	
	// add at host side for result checks
	SumArraysOnHost(h_A, h_B, hostRef, nElem);

	//execute kernle with zero copy memory
	GpuSumArrays << <grid, block >> > (h_A, h_B, h_C, nElem);

	// must be add the code before access the unified managed memory, 
	// otherwise will throw undefined exception
	hipDeviceSynchronize();

	//check device results
	CheckResults(hostRef, h_C, nElem);

	// free memory
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_C);
	
	
	free(hostRef);
	free(gpuRef);

EXIT:
	hipDeviceReset();

	system("pause");
	return 0;
}
