#include<stdio.h>
#include<hip/hip_runtime.h>
#define BLOCKNUM 16384
#define THREADNUM 512

__global__ void Ginit(int *a,int *b)
{
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    a[tid]=blockIdx.x;
    b[tid]=blockIdx.x+1;   
}

__global__ void Gmultiply(int* a,int *b)
{
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    a[tid]=a[tid]*b[tid];
} 

void CpuMul(int*a,int *b)
{
    for(int i=0;i<BLOCKNUM;i++)
	{
        for(int j=0;j<THREADNUM;j++)
		{
            int tid=i*THREADNUM+j;
            a[tid]=j;
            b[tid]=j+1;
            a[tid]=a[tid]*b[tid];
        }
    }
	printf("CPU Multiply Finished.\n");
}

int main()
{
    int* a;
    int* b;
    hipSetDevice(1);
    hipMallocManaged(&a,BLOCKNUM*THREADNUM*sizeof(int));
    hipMallocManaged(&b,BLOCKNUM*THREADNUM*sizeof(int));
    Ginit<<<BLOCKNUM,THREADNUM>>>(a,b);
	printf("GPU Initialization Finished.\n");
    hipDeviceSynchronize();
    Gmultiply<<<BLOCKNUM,THREADNUM>>>(a,b);
	printf("GPU Multiply Finished.\n");
    hipDeviceSynchronize();
    CpuMul(a,b);
    hipFree(a);
    hipFree(b);
    return 0;
} 
